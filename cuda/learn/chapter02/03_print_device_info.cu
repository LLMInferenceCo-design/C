#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<cstdio>
#include<cstdlib>
#include<cmath>

using namespace std;

#define CHECK(func)\
{\
    hipError_t error = func;\
    if(error!=hipSuccess){\
        printf("file: %s, line: %d\n",__FILE__, __LINE__);\
        printf("id: %d, reason: %s\n",error, hipGetErrorString(error));\
        exit(0);\
    }\
}\

int main(){
    int devicecount = 0;
    CHECK(hipGetDeviceCount(&devicecount)); //获得有几块GPU
    if(devicecount==0){
        cout<<"This device is pure cpu\n";
    }
    else{
        printf("There are %d GPUs\n", devicecount);
    }
    for(int dev =0;dev<devicecount;dev++){
        int driverVersion, runtimeVersion;
        hipSetDevice(dev);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,dev);//获取GPU
        printf("-----------device %d is %s----------\n", dev,prop.name);
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);

        printf("cuda driver version: %d.%d, runtime Version: %d.%d\n", \
            driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);

        printf("Total amount of global memory: %.2f GBytes, (%llu bytes)\n",\
            (float)prop.totalGlobalMem/(pow(1024.0, 3)), (unsigned long long)prop.totalGlobalMem);
        
        printf("GPU Clock rate: %.0f MHZ (%0.2f GHZ)\n", prop.clockRate*1e-3f, prop.clockRate*1e-6f);

        printf("Memory Clock rate: %.0f Mhz\n", prop.memoryClockRate*1e-3f);

        printf("Memory Bus Width: %d-bit\n", prop.memoryBusWidth);

        if(prop.l2CacheSize){
            printf("L2 Cache Size: %d Kbytes (%d bytes)\n", prop.l2CacheSize/1024, prop.l2CacheSize);
        }

        printf("Max Texture Dimension Size (x, y, z): 1D = (%d) x %d, 2D = (%d, %d) x %d\n",\
        prop.maxTexture1DLayered[0], prop.maxTexture1DLayered[1],\
        prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);

        printf("Total amount of constant memory: %lu bytes\n", prop.totalConstMem);//只读，所有SM共享

        printf("Total amount of shared memory per block: %lu Kbytes\n", prop.sharedMemPerBlock/1024);

        printf("Wrap size: %d\n", prop.warpSize);//一个wrap同时最多启动多少个thread

        printf("Maximum number of threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor); //一个SM最多多少个thread

        printf("Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);//一个block最多多少个thread

        printf("Maximum sizes of each dimension of a block: %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);

        printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n",prop.maxGridSize[0],prop.maxGridSize[1], prop.maxGridSize[2]);
        
        printf("Maximum memory pitch: %lu bytes\n", prop.memPitch);//分配二位数组时，行宽度最大值

        printf("------------------end--------------------\n");
        // cout<<warpSize<<"\n";

        hipDeviceReset();
    }
}
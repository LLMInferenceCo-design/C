#include<hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#include<iostream>
#include<chrono>
#include<cmath>

using namespace std;

#define CHECK(func)\
{\
    hipError_t error = func;\
    if(error!=hipSuccess){\
        printf("file: %s, line %d error", __FILE__, __LINE__);\
        printf("error: %d, reason: %s", error, hipGetErrorString(error));\
        exit(0);\
    }\
}

template<typename T>
void check_res(T *res1, T *res2, const int N){
    float eplos = 1.0e-8;
    for(int i=0;i<N;i++){
        if(abs(res1[i]-res2[i])>eplos){
            cout<<"gpu exe fail\n";
            exit(0);
        }
    }
    cout<<"gpu exe success\n";
}

template<typename T>
void sumOnHost(T* A, T*B, T*C, const int row, const int col){
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            C[i*col+j] = A[i*col+j]+B[i*col+j];
        }
    }
}

template<typename T>
__global__ void sumOnDevice(T*A, T*B, T*C,const int nx, const int ny,const int num){
    int idx = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    idx *= num;

    if (idx < nx * ny) {
        C=C+idx;
        A=A+idx;
        B=B+idx;
        for (int i = 0; i < num && idx + i < nx * ny; i++) {
            C[i] = A[i] + B[i];
        }
    }
}

template<typename T>
void timeOnHost(T *A, T*B, T*C, const int nx, const int ny, const int num){
    auto start = chrono::high_resolution_clock::now();
    sumOnHost(A, B, C, nx, ny);
    auto end = chrono::high_resolution_clock::now();
    auto during = chrono::duration_cast<chrono::microseconds>(end-start);
    cout<<"cpu exe time: "<<during.count()/1000<<"ms\n";
}

template<typename T>
void timeOnDevice(T*A, T*B, T*C, const int nx, const int ny, const int num){
    dim3 block(32,32);
    dim3 grid(32);
    int thread_num = (nx*ny+num-1)/num;

    grid.y = ceil((thread_num*1.0)/(block.x*block.y+grid.x));

    hipEvent_t start;
    hipEvent_t end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    sumOnDevice<T><<<grid, block>>>(A,B,C,nx,ny,num);
    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);

    float second = 0;
    hipEventElapsedTime(&second, start, end);
    cout<<"GPU exe time :"<<second<<"ms\n";
}

template<typename T>
void init_data(T*A, const int N){
    for(int i=0;i<N;i++){
        A[i]=i;
    }
}

int main(){
    int nx=1024;
    int ny =1024;
    int num=32;
    int device = 0;
    CHECK(hipSetDevice(device));

    float *A = new float[nx*ny];
    float *B = new float[nx*ny];
    float *C = new float[nx*ny];
    float *C_gpu = new float[nx*ny];

    init_data(A, nx*ny);
    init_data(B,nx*ny);

    int nbyte = nx*ny*sizeof(float);
    float *d_A,*d_B,*d_C;

    CHECK(hipMalloc((float**)&d_A, nbyte));
    CHECK(hipMalloc((float**)&d_B, nbyte));
    CHECK(hipMalloc((void**)&d_C, nbyte));

    CHECK(hipMemcpy(d_A, A, nbyte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,nbyte,hipMemcpyHostToDevice));

    timeOnHost<float>(A,B,C,nx,ny,num);
    timeOnDevice<float>(d_A, d_B, d_C, nx, ny, num);
    CHECK(hipMemcpy(C_gpu, d_C, nbyte, hipMemcpyDeviceToHost));
    check_res(C_gpu, C, nx*ny);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_gpu;
    return 0;

}
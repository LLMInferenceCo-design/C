#include<cstdio>
#include<time.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<chrono>
#include<cstdlib>


using namespace std;
#define CHECK(func) \
{ \
    const hipError_t error = func; \
    if (error!= hipSuccess){\
        printf("Error %s, %d\n", __FILE__, __LINE__);\
        printf("Error %d, reason %s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
} \

template<typename T>
void sumOnHost(T *A, T *B, T* C, const int N){
    for(int i=0;i<N;i++){
        C[i] = A[i]+B[i];
    }
}

template<typename T>
__global__ void sumOnDevice(T* A, T*B, T* C, const int N){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx<N){
        C[idx] = A[idx]+B[idx];
    }
}

template<typename T>
void funcexe_gpu(T* A, T* B, T* C, const int N, dim3 grid, dim3 block){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    sumOnDevice<T><<<grid, block>>>(A,B,C,N);

    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);

    float second=0;
    hipEventElapsedTime(&second, start, end);
    cout<<"kernel exe time: "<<second*1000<<"us\n";
    hipEventDestroy(start);
    hipEventDestroy(end);
}

template<typename T>
void funcexe_cpu(T*A, T*B, T*C, const int N){
    auto start = chrono::high_resolution_clock::now();
    sumOnHost<T>(A,B,C,N);

    auto end = chrono::high_resolution_clock::now();
    auto during = chrono::duration_cast<chrono::microseconds>(end- start);

    cout<<"cpu exe time: "<<during.count()<<"\n";

}

template<typename T>
void init_data(T* ip , const int size){
    for(int i=0;i<size;i++){
        ip[i] = i;
    }
}

template<typename T>
void checkRes(T* A, T*B, const int N){
    double epsilon = 1.0e-8;
    for(int i=0;i<N;i++){
        if(abs(A[i]-B[i])>epsilon){
            cout<<"error gpu compute\n";
            return;
        }
    }
    cout<<"gpu perfeat\n";
    return;
}

int main(){
    const int N = 1024*1024*128;
    const int block_dim = 32;
    dim3 block(block_dim);
    dim3 grid((N+block.x-1)/block.x);
    hipSetDevice(0);

    float *A_cpu = new float[N];
    float *B_cpu = new float[N];
    float *C_cpu = new float[N];
    float *C_gpu = new float[N];

    init_data(A_cpu, N);
    init_data(B_cpu,N);

    funcexe_cpu<float>(A_cpu,B_cpu,C_cpu,N);
    
    float *d_A, *d_B, *d_C;
    size_t nbyte = N* sizeof(float);
    CHECK(hipMalloc((float**)&d_A, nbyte));
    CHECK(hipMalloc((float**)&d_B, nbyte));
    CHECK(hipMalloc((float**)&d_C, nbyte));

    CHECK(hipMemcpy(d_A, A_cpu, nbyte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B_cpu, nbyte, hipMemcpyHostToDevice));

    funcexe_gpu(d_A,d_B,d_C,N,grid, block);
    // sumOnDevice<float><<<grid, block>>>(d_A, d_B, d_C, N);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(C_gpu, d_C, nbyte, hipMemcpyDeviceToHost));
    checkRes(C_cpu,C_gpu,N);
    hipDeviceReset();
    // for(int i=0;i<loop_1;i++){
    //     for(int j=0;j<loop_2;j++){
    //         cout<<C_cpu[i*loop_2+j]<<"  ";
    //     }
    //     cout<<"\n";
    // }
    return 0;

}
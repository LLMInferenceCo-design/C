#include<hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#include<cmath>
#include<iostream>

using namespace std;

#define CHECK(func)\
{\
    hipError_t error = func;\
    if(error!=hipSuccess){\
        cout<<"file: "<<__FILE__<<"  line: "<<__LINE__<<" error\n";\
        cout<<"error id: "<<error<<"  reason: "<<hipGetErrorString(error);\
        exit(0);\
    }\
}
template<typename T>
__global__ void thread_run(T* A, const int N,int num){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    idx*=num;
    A+=idx; 
    // printf("thread idx: %d\n",idx/num);
    for(int i=0;i<num && idx+i<N;i++){
        if((idx/num)%2){
            A[i]*=2;

        }
        else{
            A[i]*=4;
        }
    }
}

template<typename T>
__global__ void wrap_run(T*A, const int N, int num){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    idx*=num;
    A+=idx; 
    for(int i=0;i<num&&i+idx<N;i++){
        if(((idx/num)/ warpSize)%2==0){//常量
            A[i]*=2;
        }
        else{
            A[i]*=4;
        }
    }
}

template<typename T>
void run_time(void (*func)(T*, int, int), T*A, int N, int num, dim3 grid, dim3 block){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    (*func)<<<grid, block>>>(A,N,num);
    CHECK(hipGetLastError());  // 检查核函数启动错误
    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);

    float second =0.0;
    hipEventElapsedTime(&second, start, end);
    cout<<"func run time: "<<second<<" ms\n";
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end)); //释放时间
}

template<typename T>
void data_init(T*A, int N){
    for(int i=0;i<N;i++){
        A[i] = 1;
    }
}

template<typename T>
void print_A(T*A, int N, int num){
    for(int i=0;i<N/num;i++){
        cout<<"idx: "<<i<<"--->";
        for(int j=0;j<num;j++){
            cout<<A[i*num+j]<<" ";
        }
        cout<<"\n";
    }
}
int main(){
    int N = 64*32;
    int num = 32;
    dim3 block(32);
    dim3 grid(ceil(ceil(1.0*N/num)/block.x));

    cout<<"run start\n";

    int *A =new int [N];
    int *A_gpu = new int [N];

    data_init(A, N);

    data_init(A_gpu,N);

    int *d_A;
    int nbyte=N*sizeof(int);
    hipMalloc((void**)&d_A, nbyte);


    hipMemcpy(d_A, A, nbyte, hipMemcpyHostToDevice);
    run_time<int>(wrap_run, d_A,N,num, grid, block);
    hipMemcpy(A, d_A, nbyte, hipMemcpyDeviceToHost);

    hipMemcpy(d_A, A_gpu, nbyte, hipMemcpyHostToDevice);
    run_time<int>(thread_run, d_A, N, num, grid, block); 
    hipMemcpy(A_gpu, d_A, nbyte, hipMemcpyDeviceToHost);

    cout<<"----------------------------------------------------\n";
    print_A<int>(A, N, num);
    cout<<"----------------------------------------------------\n";
    print_A<int>(A_gpu, N,num);


    delete[] A;
    delete[] A_gpu;
    hipFree(d_A);
    return 0;
}
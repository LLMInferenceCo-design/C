
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void print(){
    printf("hello cuda\n");
}

int main(){
    printf("----------start------------\n");
    print<<<2,10>>>();
    printf("-----------end--------------\n");
    hipDeviceReset();
    return 0;
}